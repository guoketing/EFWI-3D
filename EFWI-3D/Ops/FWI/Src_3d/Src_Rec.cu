// Dongzhuo Li 02/17/2022
#include <iostream>
#include <fstream>
#include <string>
#include "Src_Rec.h"
#include "utilities.h"
#include <hip/hip_runtime.h>

#define stf(iShot,it) stf[(iShot)*(nSteps)+(it)] // row-major
// #define VERBOSE

using namespace std;
using namespace rapidjson;

Src_Rec::Src_Rec() {
	cout << "ERROR: You need to input parameter!" << endl;
	exit(1);
}

Src_Rec::Src_Rec(Parameter &para, string survey_fname, const float *stf, int group_size, const int *shot_ids) {
	string line;
	ifstream src_rec_file;

	src_rec_file.open(survey_fname);

	if (!src_rec_file.is_open()) {
		cout << "Error opening survey file" << endl;
		exit(1);
	}

	getline(src_rec_file, line);
	src_rec_file.close();

	Document json_src_rec;
	json_src_rec.Parse<0>(line.c_str());
	assert(json_src_rec.IsObject());

	int nrec = 0;
	int z_src = 10;
	int x_src = 10;
	int y_src = 10;
	int nSteps = para.nSteps();
	int nPml = para.nPoints_pml();
	float dt = para.dt();
	int *h_z_rec = nullptr;
	int *h_x_rec = nullptr;
	int *h_y_rec = nullptr;
	float *h_win_start = nullptr; // for selected window
	float *h_win_end = nullptr; // for selected window
	float *h_weights = nullptr; // for trace weighting
	float *h_source = nullptr;
	float *h_data_x = nullptr;
	float *h_data_y = nullptr;
	float *h_data_z = nullptr;
	float *h_data_obs_x = nullptr;
	float *h_data_obs_y = nullptr;
	float *h_data_obs_z = nullptr;
	float *h_res_x = nullptr;
	float *h_res_y = nullptr;
	float *h_res_z = nullptr;
	char thisShot[10];   // for shot number small than 99999
	int *d_z_rec, *d_x_rec, *d_y_rec;
	float *d_source;
	float *d_win_start, *d_win_end;
	float *d_weights;

	if_res_ = para.if_res();
	if_win_ = para.if_win();

	assert(json_src_rec.HasMember("nShots"));
	assert(json_src_rec["nShots"].IsInt());
	nShots = json_src_rec["nShots"].GetInt();

	for (int i = 0; i < group_size; i++) {
		strcpy(thisShot, ("shot" + to_string(shot_ids[i])).c_str());
		assert(json_src_rec[thisShot].HasMember("z_src"));
		assert(json_src_rec[thisShot]["z_src"].IsInt());
		z_src = json_src_rec[thisShot]["z_src"].GetInt() + nPml;
		vec_z_src.push_back(z_src);
		assert(json_src_rec[thisShot].HasMember("x_src"));
		assert(json_src_rec[thisShot]["x_src"].IsInt());
		x_src = json_src_rec[thisShot]["x_src"].GetInt() + nPml;
		vec_x_src.push_back(x_src);
		assert(json_src_rec[thisShot].HasMember("y_src"));
		assert(json_src_rec[thisShot]["y_src"].IsInt());
		y_src = json_src_rec[thisShot]["y_src"].GetInt() + nPml;
		vec_y_src.push_back(y_src);

		assert(json_src_rec[thisShot].HasMember("nrec"));
		assert(json_src_rec[thisShot]["nrec"].IsInt());
		nrec = json_src_rec[thisShot]["nrec"].GetInt();
		vec_nrec.push_back(nrec);// get the number of rec for each shot
		h_z_rec = new int[nrec];
		h_x_rec = new int[nrec];
		h_y_rec = new int[nrec];

		//read in the receiver positions for this shot
		assert(json_src_rec[thisShot].HasMember("z_rec"));
		assert(json_src_rec[thisShot]["z_rec"].IsArray());
		const Value &js_z_rec = json_src_rec[thisShot]["z_rec"];
		for (SizeType ii = 0; ii < js_z_rec.Size(); ii++) {
			h_z_rec[ii] = js_z_rec[ii].GetInt() + nPml;
			// printf("js_z_rec[%d] = %d\n", ii, js_z_rec[ii].GetInt());
		}

		assert(json_src_rec[thisShot].HasMember("x_rec"));
		assert(json_src_rec[thisShot]["x_rec"].IsArray());
		const Value &js_x_rec = json_src_rec[thisShot]["x_rec"];
		for (SizeType ii = 0; ii < js_x_rec.Size(); ii++) {
			h_x_rec[ii] = js_x_rec[ii].GetInt() + nPml;
			// printf("js_x_rec[%d] = %d\n", ii, h_x_rec[ii]);
		}

		assert(json_src_rec[thisShot].HasMember("y_rec"));
		assert(json_src_rec[thisShot]["y_rec"].IsArray());
		const Value &js_y_rec = json_src_rec[thisShot]["y_rec"];
		for (SizeType ii = 0; ii < js_y_rec.Size(); ii++) {
			h_y_rec[ii] = js_y_rec[ii].GetInt() + nPml;
			// printf("js_y_rec[%d] = %d\n", ii, h_y_rec[ii]);
		}

		// get receiver z positions for each shot
		CHECK(hipMalloc((void **)&d_z_rec, nrec * sizeof(int)));
		CHECK(hipMemcpy(d_z_rec, h_z_rec, nrec * sizeof(int), hipMemcpyHostToDevice));
		d_vec_z_rec.push_back(d_z_rec);

		// get receiver x positions for each shot
		CHECK(hipMalloc((void **)&d_x_rec, nrec * sizeof(int)));
		CHECK(hipMemcpy(d_x_rec, h_x_rec, nrec * sizeof(int), hipMemcpyHostToDevice));
		d_vec_x_rec.push_back(d_x_rec);

		// get receiver y positions for each shot
		CHECK(hipMalloc((void **)&d_y_rec, nrec * sizeof(int)));
		CHECK(hipMemcpy(d_y_rec, h_y_rec, nrec * sizeof(int), hipMemcpyHostToDevice));
		d_vec_y_rec.push_back(d_y_rec);

		// TODO:modify to use input stf
		// get the source time function for each shot
		h_source = new float[nSteps];
		for (int it = 0; it < nSteps; it++){
			h_source[it] = stf(shot_ids[i], it);
		}

		CHECK(hipMalloc((void **)&d_source, nSteps * sizeof(float)));
		CHECK(hipMemcpy(d_source, h_source, nSteps * sizeof(float), hipMemcpyHostToDevice));
		cuda_window<<<(nSteps+31)/32, 32>>>(nSteps, 1, dt, 0.001, d_source);
		CHECK(hipMemcpy(h_source, d_source, nSteps * sizeof(float), hipMemcpyDeviceToHost));
		vec_source.push_back(h_source);
		d_vec_source.push_back(d_source);
		// get the window for each shot
		if (if_win_) {
			h_win_start = new float[nrec];
			h_win_end = new float[nrec];
			assert(json_src_rec[thisShot].HasMember("win_start"));
			assert(json_src_rec[thisShot]["win_start"].IsArray());
			const Value &js_win_start = json_src_rec[thisShot]["win_start"];
			for (SizeType ii = 0; ii < js_win_start.Size(); ii++) {
				h_win_start[ii] = js_win_start[ii].GetDouble();
				// printf("h_win_start[%d] = %d\n", ii, h_win_start[ii]);
			}
			// 
			assert(json_src_rec[thisShot].HasMember("win_end"));
			assert(json_src_rec[thisShot]["win_end"].IsArray());
			const Value &js_win_end = json_src_rec[thisShot]["win_end"];
			for (SizeType ii = 0; ii < js_win_end.Size(); ii++) {
				h_win_end[ii] = js_win_end[ii].GetDouble();
				// printf("h_win_end[%d] = %d\n", ii, h_win_end[ii]);
			}
			// 
			CHECK(hipMalloc((void **)&d_win_start, nrec * sizeof(float)));
			CHECK(hipMemcpy(d_win_start, h_win_start, nrec * sizeof(float), hipMemcpyHostToDevice));
			d_vec_win_start.push_back(d_win_start);
			// 
			CHECK(hipMalloc((void **)&d_win_end, nrec * sizeof(float)));
			CHECK(hipMemcpy(d_win_end, h_win_end, nrec * sizeof(float), hipMemcpyHostToDevice));
			d_vec_win_end.push_back(d_win_end);

			delete [] h_win_start;
			delete [] h_win_end;
		}
	
		// get weights
		h_weights = new float[nrec];
		if(json_src_rec[thisShot].HasMember("weights")){
			assert(json_src_rec[thisShot]["weights"].IsArray());
			const Value &js_weights = json_src_rec[thisShot]["weights"];
			for (SizeType ii = 0; ii < js_weights.Size(); ii++) {
				h_weights[ii] = js_weights[ii].GetDouble();
				// printf("h_win_start[%d] = %d\n", ii, h_win_start[ii]);
			}
			CHECK(hipMalloc((void **)&d_weights, nrec * sizeof(float)));
			CHECK(hipMemcpy(d_weights, h_weights, nrec * sizeof(float), hipMemcpyHostToDevice));
			d_vec_weights.push_back(d_weights);
			delete [] h_weights;
		} else{
			CHECK(hipMalloc((void **)&d_weights, nrec * sizeof(float)));
			initial2DArrayGPU<<<(nrec + 32 - 1) / 32, 32>>>(d_weights, nrec, 1, 1.0);
			d_vec_weights.push_back(d_weights);
		}

		if (json_src_rec[thisShot].HasMember("src_weight")) {
			assert(json_src_rec[thisShot]["src_weight"].IsNumber());
			vec_srcweights.push_back(json_src_rec[thisShot]["src_weight"].GetDouble());
		} else{
			vec_srcweights.push_back(1.0);
		}

		hipHostAlloc((void**)&h_data_x, nSteps*nrec*sizeof(float), hipHostMallocDefault); //test
		initialArray(h_data_x, nSteps*nrec, 0.0);
		vec_data_x.push_back(h_data_x);

		hipHostAlloc((void**)&h_data_y, nSteps*nrec*sizeof(float), hipHostMallocDefault); //test
		initialArray(h_data_y, nSteps*nrec, 0.0);
		vec_data_y.push_back(h_data_y);

		hipHostAlloc((void**)&h_data_z, nSteps*nrec*sizeof(float), hipHostMallocDefault); //test
		initialArray(h_data_z, nSteps*nrec, 0.0);
		vec_data_z.push_back(h_data_z);

		if (para.if_res()) {
			// initialize the host side observed data cube
			hipHostAlloc((void**)&h_data_obs_x, nSteps*nrec*sizeof(float), hipHostMallocDefault);
			initialArray(h_data_obs_x, nSteps*nrec, 0.0);
			vec_data_obs_x.push_back(h_data_obs_x);
			// initialize the host side data residual
			hipHostAlloc((void**)&h_res_x, nSteps*nrec*sizeof(float), hipHostMallocDefault); //test
			initialArray(h_res_x, nSteps*nrec, 0.0);
			vec_res_x.push_back(h_res_x);

			// initialize the host side observed data cube
			hipHostAlloc((void**)&h_data_obs_y, nSteps*nrec*sizeof(float), hipHostMallocDefault);
			initialArray(h_data_obs_y, nSteps*nrec, 0.0);
			vec_data_obs_y.push_back(h_data_obs_y);
			// initialize the host side data residual
			hipHostAlloc((void**)&h_res_y, nSteps*nrec*sizeof(float), hipHostMallocDefault); //test
			initialArray(h_res_y, nSteps*nrec, 0.0);
			vec_res_y.push_back(h_res_y);

			// initialize the host side observed data cube
			hipHostAlloc((void**)&h_data_obs_z, nSteps*nrec*sizeof(float), hipHostMallocDefault);
			initialArray(h_data_obs_z, nSteps*nrec, 0.0);
			vec_data_obs_z.push_back(h_data_obs_z);
			// initialize the host side data residual
			hipHostAlloc((void**)&h_res_z, nSteps*nrec*sizeof(float), hipHostMallocDefault); //test
			initialArray(h_res_z, nSteps*nrec, 0.0);
			vec_res_z.push_back(h_res_z);
		}
	}
}


Src_Rec::~Src_Rec() {
	for(int i=0; i<d_vec_x_rec.size(); i++) {
		CHECK(hipFree(d_vec_z_rec.at(i)));
		CHECK(hipFree(d_vec_x_rec.at(i)));
		CHECK(hipFree(d_vec_y_rec.at(i)));
	}

	for(int i=0; i<vec_source.size(); i++) {
		delete [] vec_source.at(i);
		CHECK(hipFree(d_vec_source.at(i)));
	}

	for(int i=0; i<vec_data_x.size(); i++) {
		CHECK(hipHostFree(vec_data_x.at(i))); //test
		CHECK(hipHostFree(vec_data_y.at(i))); //test
		CHECK(hipHostFree(vec_data_z.at(i))); //test
	}

	if (if_res_) {
		for(int i=0; i<vec_data_obs_x.size(); i++) {
			CHECK(hipHostFree(vec_data_obs_x.at(i)));
			CHECK(hipHostFree(vec_data_obs_y.at(i)));
			CHECK(hipHostFree(vec_data_obs_z.at(i)));
			CHECK(hipHostFree(vec_res_x.at(i)));
			CHECK(hipHostFree(vec_res_y.at(i)));
			CHECK(hipHostFree(vec_res_z.at(i)));
		}
	}

	if (if_win_) {
		for(int i=0; i<d_vec_win_start.size(); i++) {
			CHECK(hipFree(d_vec_win_start.at(i)));
			CHECK(hipFree(d_vec_win_end.at(i)));
		}
	}

	for(int i=0; i<d_vec_weights.size(); i++) {
		CHECK(hipFree(d_vec_weights.at(i)));
	}
}